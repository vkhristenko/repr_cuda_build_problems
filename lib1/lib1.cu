
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>

__device__
void some_dev_func() {}

__global__
void kernel_test() { 
    printf("hello from lib1 kernel\n"); 
    some_dev_func();
}

void doEntryPoint() { 
    std::cout << "hello from lib1\n"; 
    auto check_error = [](auto code) {
        if (code != hipSuccess) {
            std::cout << hipGetErrorString(code) << std::endl;
            assert(false);
        }
    };

    kernel_test<<<1,1>>>();
    check_error(hipGetLastError());
    hipDeviceSynchronize();
}

extern "C" {
    void entryPoint() {
        doEntryPoint();
    }
}

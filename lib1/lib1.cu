#include "hip/hip_runtime.h"
#include <iostream>

__global__
void kernelt_test() { printf("hello from lib1 kernel\n"); }

extern "C" {
    void entryPoint() { 
        auto check_error = [](auto code) {
            if (code != hipSuccess) {
                std::cout << hipGetErrorString(code) << std::endl;
                assert(false);
            }
        };

        std::cout << "hello from lib1\n"; 
        kernel_test<<<1,1>>>();
        check_error(hipGetLastError());
        hipDeviceSynchronize();
    }
}

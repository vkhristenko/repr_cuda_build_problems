#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>

#include "kernels.h"

namespace lib2 {

__device__
void some_dev_func() {}

__global__
void kernel_test() { 
    printf("hello from lib2 kernel\n"); 
    some_dev_func();
}

void doEntryPoint() { 
    std::cout << "hello from lib2\n"; 
    auto check_error = [](auto code) {
        if (code != hipSuccess) {
            std::cout << hipGetErrorString(code) << std::endl;
            assert(false);
        }
    };

    kernel_test<<<1,1>>>();
    check_error(hipGetLastError());
    hipDeviceSynchronize();
}

}
